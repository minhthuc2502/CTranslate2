#include "hip/hip_runtime.h"
#include "ctranslate2/ops/layer_norm.h"

#include "cuda/helpers.h"
#include "cuda/utils.h"

namespace at {
  namespace native {

    // Forward declaration of the CUDA kernels.
    template <typename T, typename SizeT>
    __global__ void LayerNormForwardCUDAKernel(SizeT N,
                                               float eps,
                                               const T* X,
                                               T* Y);

  }
}

namespace ctranslate2 {
  namespace ops {

#define CUDA_NUM_THREADS 512

    template <Device D, typename T>
    void LayerNormWMask::compute(const StorageView& input,
                            const dim_t axis,
                            const dim_t outer_size,
                            const dim_t axis_size,
                            const dim_t,
                            StorageView& output) const {
      at::native::LayerNormForwardCUDAKernel<cuda::device_type<T>, cuda::index_t>
        <<<outer_size, CUDA_NUM_THREADS, 0, cuda::get_cuda_stream()>>>(
          axis_size,
          _epsilon,
          cuda::device_cast(input.data<T>()),
          cuda::device_cast(output.data<T>()));
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    LayerNormWMask::compute<Device::CUDA, T>(const StorageView& input,       \
                                        const dim_t axis,               \
                                        const dim_t outer_size,         \
                                        const dim_t axis_size,          \
                                        const dim_t inner_size,         \
                                        StorageView& output) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)
    DECLARE_IMPL(bfloat16_t)

  }
}

#include <cub/block/block_reduce.cuh>

namespace at {
  namespace native {

    template <typename T, typename SizeT>
    __global__ void LayerNormForwardCUDAKernel(SizeT N,
                                               float eps,
                                               const T* X,
                                               T* Y) {
      typedef hipcub::BlockReduce<float, CUDA_NUM_THREADS> BlockReduce;
      __shared__ typename BlockReduce::TempStorage m_temp_storage;
      __shared__ typename BlockReduce::TempStorage v_temp_storage;
      __shared__ float s_mean;
      __shared__ float s_variance;

      const SizeT i = blockIdx.x;

      float num_elements_not_masked = 0;
      float sum1 = 0;
      float sum2 = 0;
      for (SizeT j = threadIdx.x; j < N; j += blockDim.x) {
        const SizeT index = i * N + j;
        sum1 += float(X[index]);
        sum2 += float(X[index]) * float(X[index]);
        num_elements_not_masked += 1;
      }
      sum1 = BlockReduce(m_temp_storage).Sum(sum1);
      sum2 = BlockReduce(v_temp_storage).Sum(sum2);
      num_elements_not_masked = BlockReduce(v_temp_storage).Sum(num_elements_not_masked);
      if (threadIdx.x == 0) {
        sum1 /= num_elements_not_masked;
      }
      for (SizeT j = threadIdx.x; j < N; j += blockDim.x) {
        const SizeT index = i * N + j;
        sum2 += (float(X[index]) - s_mean) * (float(X[index]) - s_mean);
      }
      sum2 = BlockReduce(v_temp_storage).Sum(sum2);
      if (threadIdx.x == 0) {
        sum2 /= num_elements_not_masked;
        s_mean = sum1;
        s_variance = rsqrtf(sum2 + eps);
      }

      __syncthreads();

      for (SizeT j = threadIdx.x; j < N; j += blockDim.x) {
        const SizeT index = i * N + j;
        Y[index] = (float(X[index]) - s_mean) * s_variance;
      }
    }

  }
}
